
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include<assert.h>

__global__ void add(float *a, float *b, float *c)
{

int index = threadIdx.x + blockIdx.x * blockDim.x;
c[index] = a[index] + b[index];


}



int main(void)
{





float *a, *b, *c;
float *d_a, *d_b, *d_c,stop,start,realTime;
int i;
int size = 250000*sizeof(float);



hipMalloc((void **)&d_a, size);
hipMalloc((void **)&d_b, size);
hipMalloc((void **)&d_c, size);

a = (float*)malloc(size); 
b = (float*)malloc(size);
c = (float*)malloc(size); 

assert((start = clock())!=-1);			//starting real time



for(i=0;i<250000;i++)
{
	a[i] = 2;
	b[i] = 7;
}


hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

add<<<500,500>>>(d_a, d_b, d_c);


hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);




for(i=0;i<250000;i++)
{
printf("%e\n",c[i]);
printf("%e\n",a[i]);
printf("%e\n",b[i]);	
}

stop = clock();
	  realTime = (double) (stop-start)/CLOCKS_PER_SEC;
          printf("Run time: %2.2f seconds \n", realTime);	

hipFree(d_a); hipFree(d_b); hipFree(d_c); 


free(a);
free(b);
free(c);

return 0;

}

