#include<stdio.h>
#include<hipsparse.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include "/home/suhas/Desktop/myNavierStokesCUDASolver/sourceCode/solver/prepareAndCompute.cu"
#include "/home/suhas/Desktop/myNavierStokesCUDASolver/sourceCode/solver/create.c"
#include "/home/suhas/Desktop/myNavierStokesCUDASolver/sourceCode/solver/clear.c"



void allocate()
{

	

for(i=0;i<n;i++) 
{
	  if(i== 0 )
	  {
		  lower[i] = -0.0001996805;
		  upper[i]= -0.0001996805;
		  middle[i] = 1.0;
		  c[i] = 0;
	  }
	  if(i== n-1)
	  {
		 lower[i]=	 -0.0001996805 ;
		 upper[i] = -0.0001996805;
		 middle[i]  = 1.0;
		 c[i] = 0.0003993610;
	  }	
	  if(i>0 && i<n-1)
	  {
		  lower[i] =-0.0001996805;
		  middle[i] = 1.0;
		  upper[i] = -0.0001996805;
		  c[i] = 0;
	  }


	
}	
prepareAndCompute();



}




int main()
{
int j;

n=20;
 size = sizeof(float) *n;

create();


for(j=0;j<200;j++)
 allocate();	



clear();


   
return 0;   
}


