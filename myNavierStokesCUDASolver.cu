#include "hip/hip_runtime.h"
//my Navier Stokes solution


#include<stdio.h>
#include<assert.h>
#include<time.h>
#include "preProcessing/input/zeroTimeValues.c"
#include "preProcessing/input/input.c"
#include "preProcessing/mesh/boundaryConditions.c"
#include "sourceCode/algorithm/pressureVelocityCoupling/simple.c"
#include "sourceCode/control/solverControls/shift.c"
#include "sourceCode/algorithm/velocity/makeVelocityVectors.c"
#include "sourceCode/control/outputControls/writeOutput.c"
#include "sourceCode/control/outputControls/writeMesh.c"
#include "sourceCode/control/solverControls/incrementTime.c"
#include "sourceCode/control/outputControls/courantNumber.c"


                 
int main()
{

	defaultInput(); 	   	//Inputting the values required

	dynamicAllocate();		//dynamically allocates the arrays

	zeroTimeValues();   	//Gives initial values to all the variables and fields at time 0
	
	writeMesh();	      	//writes internal mesh to mesh.csv 
	
	


	t=1;			//output file starts from 1st time step
	simulationTime=dt;		//time counting starts from dt
	
	assert((start = clock())!=-1);			//starting real time

	
	while(simulationTime<=endTime)         //time loop                                                        
 	{
	  l = 1;
	  maxMassResidual = 100;

	  printf("\n\n\nsimulationTime :%lf seconds\n",simulationTime);
	
	  boundaryConditions();     //updates the values of the boundary cells
	
	  ghostCells();
	
	  simple(); 		    //simple algorithm

	  makeVelocityVectors();    //Interpolation of the velocity vectors back from staggerd cell centers to the pressure cell centers 

	  writeOutput();            //Outputs values to files

	  shift();		    //shifts the values to l=0
	  
	  courantNumber();	    //calculates courant number
	
	  incrementTime();	    //increments time variables

	
	  stop = clock();
	  realTime = (double) (stop-start)/CLOCKS_PER_SEC;
          printf("Run time: %2.2f seconds \n", realTime);				//printing real time
	}
  
return 0;	
}
 










