//prepares GPU and solve the matrix


#include<stdio.h>
#include<hipsparse.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include "/home/suhas/Desktop/myNavierStokesCUDASolver/preProcessing/input/variables.H"

 
void prepareAndCompute()
{


 	 /* Initialize cuSPARSE library */
  	hipsparseStatus_t status;
  	hipsparseHandle_t handle=0;
  	status= hipsparseCreate(&handle);


   /* Copy data into device from host for TDMA/ another process*/
  hipMemcpy(d_upper,upper,size,hipMemcpyHostToDevice);
  hipMemcpy(d_middle,middle,size,hipMemcpyHostToDevice);
  hipMemcpy(d_lower,lower,size,hipMemcpyHostToDevice);
  hipMemcpy(d_c,c,size,hipMemcpyHostToDevice);
 
   /*TDMA solver using the cuSPARSE library*/
   status = cusparseSgtsv(handle, n, 1,d_lower, d_middle, d_upper, d_c, n) ;
   if (status != HIPSPARSE_STATUS_SUCCESS)
	{													// vimp step for using status and for debugging!!!
        printf("CUSPARSE Library initialization failed");
    
    }
  hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);	   // copy back results as the output is returned in R_d vector

													  // which is also the RHS of AX=R . 

  
   
}


